#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>

#include "restrain.h"
#include "main/defines.h"
#include "system/system.h"
#include "system/state.h"
#include "run/run.h"
#include "system/potential.h"

#include "main/real3.h"

/* resd raafik 05-24-2025 */ 
template <bool flagBox,typename box_type>
__global__ void getforce_resd_kernel(int resdCount,struct ResdPotential *resds,real3 *position,real3_f *force,box_type box,real_e *energy)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  ResdPotential resdp;
  real r1, r2, delref;
  real3 dr1; 
  real3 dr2;
  real fresd=0;
  real lEnergy=0;
  extern __shared__ real sEnergy[];
  real3 xi1, xi2, xj1, xj2;

  if (i<resdCount) {
    resdp=resds[i];
    xi1=position[resdp.i1];
    xi2=position[resdp.i2];
    xj1=position[resdp.j1];
    xj2=position[resdp.j2];
    dr1=real3_subpbc<flagBox>(xi1,xi2,box);
    dr2=real3_subpbc<flagBox>(xj1,xj2,box);
    r1=real3_mag<real>(dr1);
    r2=real3_mag<real>(dr2);
    delref=resdp.ci*r1+resdp.cj*r2-resdp.rdist;
    fresd=delref*resdp.kdist;
    if (energy) lEnergy=((real)0.5)*fresd*delref;

    // Pair 1: (i1, i2)
    at_real3_scaleinc(&force[resdp.i1],  fresd * resdp.ci / r1, dr1);
    at_real3_scaleinc(&force[resdp.i2], -fresd * resdp.ci / r1, dr1);

    // Pair 2: (j1, j2)
    at_real3_scaleinc(&force[resdp.j1],  fresd * resdp.cj / r2, dr2);
    at_real3_scaleinc(&force[resdp.j2], -fresd * resdp.cj / r2, dr2);
  }
    // Energy, if requested
  if (energy) {
    real_sum_reduce(lEnergy,sEnergy,energy);
  }
}

template <bool flagBox,typename box_type>
void getforce_resdT(System *system,box_type box,bool calcEnergy)
{
  Potential *p=system->potential;
  State *s=system->state;
  Run *r=system->run;
  int N;
  int shMem=0;
  real_e *pEnergy=NULL;
  if (r->calcTermFlag[eebias]==false) return; //eeresd

  if (calcEnergy) {
    shMem=BLBO*sizeof(real)/32;
    pEnergy=s->energy_d+eebias; // eeresd
  }
  N=p->resdCount;
  if (N>0) getforce_resd_kernel<flagBox><<<(N+BLBO-1)/BLBO,BLBO,shMem,r->biaspotStream>>>(N,p->resds_d,(real3*)s->position_fd,(real3_f*)s->force_d,box,pEnergy);
}

void getforce_resd(System *system,bool calcEnergy)
{
  if (system->state->typeBox) {
    getforce_resdT<true>(system,system->state->tricBox_f,calcEnergy);
  } else {
    getforce_resdT<false>(system,system->state->orthBox_f,calcEnergy);
  }
}
/* resd raafik 05-24-2025 */

template <bool flagBox,typename box_type>
__global__ void getforce_noe_kernel(int noeCount,struct NoePotential *noes,real3 *position,real3_f *force,box_type box,real_e *energy)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  NoePotential noep;
  real r,r_r0;
  real3 dr;
  real fnoe=0;
  real lEnergy=0;
  extern __shared__ real sEnergy[];
  real3 xi,xj;
  
  if (i<noeCount) {
    // Geometry
    noep=noes[i];
    xi=position[noep.i];
    xj=position[noep.j];
    dr=real3_subpbc<flagBox>(xi,xj,box);
    r=real3_mag<real>(dr);
    if (r<noep.rmin) {
      r_r0=r-noep.rmin;
      fnoe=noep.kmin*r_r0;
      if (energy) lEnergy=((real)0.5)*fnoe*r_r0;
    } else if (r>noep.rmax) {
      r_r0=r-noep.rmax;
      if (noep.rswitch>0 && r_r0>noep.rswitch) {
        real bswitch=(noep.rpeak-noep.rswitch)/noep.nswitch*pow(noep.rswitch,noep.nswitch+1);
        real aswitch=0.5*noep.rswitch*noep.rswitch-noep.rpeak*noep.rswitch-noep.rswitch*(noep.rpeak-noep.rswitch)/noep.nswitch;
        fnoe=noep.kmax*(noep.rpeak-bswitch*pow(r_r0,-noep.nswitch-1));
        if (energy) lEnergy=noep.kmax*(aswitch+bswitch*pow(r_r0,-noep.nswitch)+noep.rpeak*r_r0);
      } else {
        fnoe=noep.kmax*r_r0;
        if (energy) lEnergy=((real)0.5)*fnoe*r_r0;
      }
    }
    // Spatial force
    at_real3_scaleinc(&force[noep.i], fnoe/r,dr);
    at_real3_scaleinc(&force[noep.j],-fnoe/r,dr);
  }

  // Energy, if requested
  if (energy) {
    real_sum_reduce(lEnergy,sEnergy,energy);
  }
}

template <bool flagBox,typename box_type>
void getforce_noeT(System *system,box_type box,bool calcEnergy)
{
  Potential *p=system->potential;
  State *s=system->state;
  Run *r=system->run;
  int N;
  int shMem=0;
  real_e *pEnergy=NULL;

  if (r->calcTermFlag[eebias]==false) return;

  if (calcEnergy) {
    shMem=BLBO*sizeof(real)/32;
    pEnergy=s->energy_d+eebias;
  }

  N=p->noeCount;
  if (N>0) getforce_noe_kernel<flagBox><<<(N+BLBO-1)/BLBO,BLBO,shMem,r->biaspotStream>>>(N,p->noes_d,(real3*)s->position_fd,(real3_f*)s->force_d,box,pEnergy);
}

void getforce_noe(System *system,bool calcEnergy)
{
  if (system->state->typeBox) {
    getforce_noeT<true>(system,system->state->tricBox_f,calcEnergy);
  } else {
    getforce_noeT<false>(system,system->state->orthBox_f,calcEnergy);
  }
}

template <bool flagBox,typename box_type>
__global__ void getforce_harm_kernel(int harmCount,struct HarmonicPotential *harms,real3 *position,real3_f *force,box_type box,real_e *energy)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int ii;
  real r2;
  real3 dr;
  HarmonicPotential hp;
  real krnm2;
  real lEnergy=0;
  extern __shared__ real sEnergy[];
  real3 xi,x0;
  
  if (i<harmCount) {
    // Geometry
    hp=harms[i];
    ii=hp.idx;
    xi=position[ii];
    x0=hp.r0;
// NOTE #warning "Unprotected division"
    dr=real3_subpbc<flagBox>(xi,x0,box);
    r2=real3_mag2<real>(dr);
    krnm2=(r2 ? (hp.k*pow(r2,((real)0.5)*hp.n-1)) : 0); // NaN guard it
    
    if (energy) {
      lEnergy=krnm2*r2;
    }
    at_real3_scaleinc(&force[ii], hp.n*krnm2,dr);
  }

  // Energy, if requested
  if (energy) {
    real_sum_reduce(lEnergy,sEnergy,energy);
  }
}

template <bool flagBox,typename box_type>
void getforce_harmT(System *system,box_type box,bool calcEnergy)
{
  Potential *p=system->potential;
  State *s=system->state;
  Run *r=system->run;
  int N;
  int shMem=0;
  real_e *pEnergy=NULL;

  if (r->calcTermFlag[eebias]==false) return;

  if (calcEnergy) {
    shMem=BLBO*sizeof(real)/32;
    pEnergy=s->energy_d+eebias;
  }

  N=p->harmCount;
  if (N>0) getforce_harm_kernel<flagBox><<<(N+BLBO-1)/BLBO,BLBO,shMem,r->biaspotStream>>>(N,p->harms_d,(real3*)s->position_fd,(real3_f*)s->force_d,box,pEnergy);
}

void getforce_harm(System *system,bool calcEnergy)
{
  if (system->state->typeBox) {
    getforce_harmT<true>(system,system->state->tricBox_f,calcEnergy);
  } else {
    getforce_harmT<false>(system,system->state->orthBox_f,calcEnergy);
  }
}
